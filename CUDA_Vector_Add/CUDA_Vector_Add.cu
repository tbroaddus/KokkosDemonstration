#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 256

__global__ void vector_add(double *a, double *b, double *c) {
  if (threadIdx.x < N) 
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x]; 
}

int main() {
  double *a, *b, *c;
  int size = N * sizeof(double)

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);


  a = (double*)malloc(size);
  b = (double*)malloc(size);
  c = (double*)malloc(size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_a, b, size, hipMemcpyHostToDevice);

  vector_add<<<1,N>>>(d_a, d_b, d_c);

  hipMemcpy(d_c, c, size, hipMemcpyDeviceToHost);

  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
